#include "LeNet5_cuda.h"

void LeNet5_cuda::predict(int batch) {
    // TODO: Implement conv1
    // TODO: Implement relu
    // TODO: Implement pool1
    // TODO: Implement conv2
    // TODO: Implement relu
    // TODO: Implement pool2
    // TODO: Implement fc1
    // TODO: Implement relu
    // TODO: Implement fc2
    // TODO: Implement relu
    // TODO: Implement fc3

    /* NOTE: unless you want to make a major change to this class structure, 
    *  you need to write your output to the device memory d_output 
    *  so that classify() can handle the rest.
    */
}

void LeNet5_cuda::prepare_device_memory(uint8_t* image) {
  // Alloc Model Parameters
  hipMalloc((void**)&d_conv1_weight,
             sizeof(double) * conv1_in_channel * conv1_out_channel *
                 conv1_kernel_size * conv1_kernel_size);
  hipMalloc((void**)&d_conv1_bias, sizeof(double) * conv1_out_channel);
  hipMalloc((void**)&d_conv2_weight,
             sizeof(double) * conv2_in_channel * conv2_out_channel *
                 conv2_kernel_size * conv2_kernel_size);
  hipMalloc((void**)&d_conv2_bias, sizeof(double) * conv2_out_channel);
  hipMalloc((void**)&d_fc1_weight,
             sizeof(double) * fc1_in_channel * fc1_out_channel);
  hipMalloc((void**)&d_fc1_bias, sizeof(double) * fc1_out_channel);
  hipMalloc((void**)&d_fc2_weight,
             sizeof(double) * fc2_in_channel * fc2_out_channel);
  hipMalloc((void**)&d_fc2_bias, sizeof(double) * fc2_out_channel);
  hipMalloc((void**)&d_fc3_weight,
             sizeof(double) * fc3_in_channel * fc3_out_channel);
  hipMalloc((void**)&d_fc3_bias, sizeof(double) * fc3_out_channel);

  // Alloc Activations
  hipMalloc((void**)&d_image,
             sizeof(uint8_t) * batch * input_size * input_size * input_channel);
  hipMalloc((void**)&d_input,
             sizeof(double) * batch * input_channel * input_size * input_size);
  hipMalloc((void**)&d_C1_feature_map,
             sizeof(double) * batch * C1_channel * C1_size * C1_size);
  hipMalloc((void**)&d_S2_feature_map,
             sizeof(double) * batch * S2_channel * S2_size * S2_size);
  hipMalloc((void**)&d_C3_feature_map,
             sizeof(double) * batch * C3_channel * C3_size * C3_size);
  hipMalloc((void**)&d_S4_feature_map,
             sizeof(double) * batch * S4_channel * S4_size * S4_size);
  hipMalloc((void**)&d_C5_layer, sizeof(double) * batch * C5_size);
  hipMalloc((void**)&d_F6_layer, sizeof(double) * batch * F6_size);
  hipMalloc((void**)&d_output, sizeof(double) * batch * output_size);

  // Copy Parameters
  hipMemcpy(d_conv1_weight, conv1_weight,
             sizeof(double) * conv1_in_channel * conv1_out_channel *
                 conv1_kernel_size * conv1_kernel_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv1_bias, conv1_bias, sizeof(double) * conv1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv2_weight, conv2_weight,
             sizeof(double) * conv2_in_channel * conv2_out_channel *
                 conv2_kernel_size * conv2_kernel_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv2_bias, conv2_bias, sizeof(double) * conv2_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc1_weight, fc1_weight,
             sizeof(double) * fc1_in_channel * fc1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc1_bias, fc1_bias, sizeof(double) * fc1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc2_weight, fc2_weight,
             sizeof(double) * fc2_in_channel * fc2_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc2_bias, fc2_bias, sizeof(double) * fc2_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc3_weight, fc3_weight,
             sizeof(double) * fc3_in_channel * fc3_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc3_bias, fc3_bias, sizeof(double) * fc3_out_channel,
             hipMemcpyHostToDevice);
  // copy input image
  size_t image_size = batch * input_size * input_size * input_channel;
  hipMemcpy(d_image, image, image_size * sizeof(uint8_t),
             hipMemcpyHostToDevice);
}

void LeNet5_cuda::classify(int* predict, int batch) {
  // read logits back to cpu
  hipMemcpy(output, d_output, sizeof(double) * output_size * batch,
             hipMemcpyDeviceToHost);
  // Softmax
  softmax(output, predict, batch, output_size);
}

LeNet5_cuda::~LeNet5_cuda() {
  hipFree(d_conv1_weight);   
  hipFree(d_conv2_weight);   
  hipFree(d_conv1_bias);     
  hipFree(d_conv2_bias);     
  hipFree(d_fc1_weight);     
  hipFree(d_fc2_weight);     
  hipFree(d_fc3_weight);     
  hipFree(d_fc1_bias);       
  hipFree(d_fc2_bias);       
  hipFree(d_fc3_bias);       

  hipFree(d_image);          
  hipFree(d_input);          
  hipFree(d_C1_feature_map); 
  hipFree(d_S2_feature_map); 
  hipFree(d_C3_feature_map); 
  hipFree(d_S4_feature_map); 
  hipFree(d_C5_layer);      
  hipFree(d_F6_layer);     
  hipFree(d_output);       
  hipFree(d_predict_cuda);   
}
