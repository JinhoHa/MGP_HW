#include "hip/hip_runtime.h"
#include "LeNet5_cuda.h"

__global__
void normalize(uint8_t* image, double* input) {
  // Initialize variables
  // double max_int = 255.0L;
  // double mean = 0.5L;
  // double var = 0.5L;
  // // Normalize
  // for (int i = 0; i < batch * input_channel * input_size * input_size; i++) {
  //   input[i] = image[i] / max_int;       // transforms.ToTensor();
  //   input[i] = (input[i] - mean) / var;  // transforms.Normalize();
  // }
  // cuda
  // blockIdx.y : batch, blockIdx.x : Channel
  // threadIdx.y : input_size, threadIdx.x : input_size
  // int taskIdx = blockIdx.y * gridDim.x * blockDim.y * blockDim.x
  //               + blockIdx.x * blockDim.y * blockDim.x
  //               + threadIdx.y * blockDim.x
  //               + threadIdx.x;
  int taskIdx = blockIdx.x * blockDim.x + threadIdx.x;
  input[taskIdx] = image[taskIdx] / double(255.0);
  input[taskIdx] = (input[taskIdx] - 0.5) / 0.5;
}

__global__
void cuda_conv(double* input, double* output, double* weight,
                      double* bias, int H, int W, int IC, int K) {
    // blockIdx.y : mini-batch (b)
    // blockIdx.x : output Channel (oc), gridDim.x : OC
    // threadIdx.y : Height (h), blockDim.y : H_OUT
    // threadIdx.x : Width (w), blockDim.x : W_OUT
    int b = blockIdx.y;
    int oc = blockIdx.x;
    int h = threadIdx.y;
    int w = threadIdx.x;
    int OC = gridDim.x;
    int H_OUT = blockDim.y;
    int W_OUT = blockDim.x;
    // int taskIdx = blockIdx.y * gridDim.x * blockDim.y * blockDim.x
    //               + blockIdx.x * blockDim.y * blockDim.x
    //               + threadIdx.y * blockDim.x
    //               + threadIdx.x;
    int BLKSIZE = H_OUT * W_OUT;
    // int taskIdx = b * OC * BLKSIZE + oc * BLKSIZE + h * W_OUT + w;

    double val = bias[oc];
    for (int ic=0; ic<IC; ic++) {
      int input_base = b * (IC * H * W) + ic * (H * W)
                       + h * (W) + w;
      // int input_base = fma(b, IC, ic) * (H * W) + fma(h, W, w);
      int kernel_base = oc * (IC * K * K) + ic * (K * K);
      // int kernel_base = fma(oc, IC, ic) * (K * K);
      for (int kh = 0; kh < K; kh++)
        for (int kw = 0; kw < K; kw++) {
          val += input[input_base + kh * (W) + kw] *
                 weight[kernel_base + kh * (K) + kw];
          // val += input[input_base + fma(kh, W, kw)] *
          //        weight[kernel_base + fma(kh, K, kw)];
        }
    }
    output[b * OC * BLKSIZE + oc * BLKSIZE + h * W_OUT + w] = val;
    // output[fma(b, OC, oc) * H_OUT * W_OUT + fma(h, W_OUT, w)] = val;
}

__global__
void cuda_conv1(double* input, double* output, double* weight,
                      double* bias) {
    // blockIdx.y : mini-batch (b)
    // blockIdx.x : output Channel (oc), gridDim.x : OC
    // threadIdx.y : Height (h), blockDim.y : H_OUT
    // threadIdx.x : Width (w), blockDim.x : W_OUT
    int b = blockIdx.y;
    int oc = blockIdx.x;
    int h = threadIdx.y;
    int w = threadIdx.x;
    int OC = gridDim.x;
    int H = 32;
    int W = 32;
    int H_OUT = 28;
    int W_OUT = 28;
    int IC = 3;
    int K = 5;

    double val = bias[oc];
    #pragma unroll
    for (int ic=0; ic<3; ic++) {
      int input_base = b * (IC * H * W) + ic * (H * W)
                       + h * (W) + w;
      int kernel_base = oc * (IC * K * K) + ic * (K * K);
      for (int kh = 0; kh < K; kh++)
        for (int kw = 0; kw < K; kw++) {
          val += input[input_base + kh * (W) + kw] *
                 weight[kernel_base + kh * (K) + kw];
        }
    }

    // if(w < W_OUT) {
    //   int outBlockSize = H_OUT * W_OUT;
    //   output[b * OC * outBlockSize + oc * outBlockSize + h * W_OUT + w] = val;
    // }
    int outBlockSize = H_OUT * W_OUT;
    output[b * OC * outBlockSize + oc * outBlockSize + h * W_OUT + w] = val;
}

__global__
void cuda_conv2(double* input, double* output, double* weight,
                      double* bias) {
    // blockIdx.y : mini-batch (b)
    // blockIdx.x : output Channel (oc), gridDim.x : OC
    // threadIdx.y : Height (h), blockDim.y : H_OUT
    // threadIdx.x : Width (w), blockDim.x : W_OUT
    int b = blockIdx.y;
    int oc = blockIdx.x;
    int h = threadIdx.y;
    int w = threadIdx.x;
    int OC = gridDim.x;
    int H = 14;
    int W = 14;
    int H_OUT = 10;
    int W_OUT = 10;
    int IC = 6;
    int K = 5;

    double val = bias[oc];
    #pragma unroll
    for (int ic=0; ic<6; ic++) {
      int input_base = b * (IC * H * W) + ic * (H * W)
                       + h * (W) + w;
      int kernel_base = oc * (IC * K * K) + ic * (K * K);
      for (int kh = 0; kh < K; kh++)
        for (int kw = 0; kw < K; kw++) {
          val += input[input_base + kh * (W) + kw] *
                 weight[kernel_base + kh * (K) + kw];
        }
    }

    // if(w < W_OUT) {
    //   int outBlockSize = H_OUT * W_OUT;
    //   output[b * OC * outBlockSize + oc * outBlockSize + h * W_OUT + w] = val;
    // }
    int outBlockSize = H_OUT * W_OUT;
    output[b * OC * outBlockSize + oc * outBlockSize + h * W_OUT + w] = val;
}

// shared memory
__global__
void cuda_conv1_sh(double* input, double* output, double* weight,
                      double* bias, int B, int H, int W, int IC, int OC,
                      int K) {

    int taskIdx = blockIdx.y * gridDim.x * blockDim.y * blockDim.x
                  + blockIdx.x * blockDim.y * blockDim.x
                  + threadIdx.y * blockDim.x
                  + threadIdx.x;

    __shared__ double input_sh[32][32];
    __shared__ double weight_sh[5][5];
    double val = bias[blockIdx.x];
    for (int ic = 0; ic < IC; ic++) {
      // load weights to shared memory
      if(threadIdx.y < K && threadIdx.x < K) {
        weight_sh[threadIdx.y][threadIdx.x] = weight[
          blockIdx.x * IC * K * K + ic * K * K + threadIdx.y * K + threadIdx.x];
      }
      __syncthreads();

      // load input to shared memory
      int input_base = blockIdx.y * (IC * H * W) + ic * (H * W);
      for(int ih = threadIdx.y; ih < H; ih += blockDim.y) {
        for(int iw = threadIdx.x; iw < W; iw += blockDim.x) {
          input_sh[ih][iw] = input[input_base + ih * W + iw];
        }
      }
      __syncthreads();

      for (int kh = 0; kh < K; kh++) {
        for (int kw = 0; kw < K; kw++) {
          val += input_sh[threadIdx.y + kh][threadIdx.x + kw] *
                 weight_sh[kh][kw];
        }
      }
      __syncthreads();
    }
    output[taskIdx] = val;
}

// shared memory + (32*32) thread block
__global__
void cuda_conv_sh32(double* input, double* output, double* weight,
                      double* bias, int B, int H, int W, int IC, int OC,
                      int K) {
    // // blockIdx.y : mini-batch (b)
    // // blockIdx.x : output Channel (oc)
    // // threadIdx.y : Height (h), BlockDim.y : H
    // // threadIdx.x : Width (w), BlockDim.x : W
    // int taskIdx = blockIdx.y * gridDim.x * blockDim.y * blockDim.x
    //               + blockIdx.x * blockDim.y * blockDim.x
    //               + threadIdx.y * blockDim.x
    //               + threadIdx.x;
    // double val = bias[blockIdx.x];
    // for (int ic=0; ic<IC; ic++) {
    //   int input_base = blockIdx.y * (IC * H * W) + ic * (H * W)
    //                    + threadIdx.y * (W) + threadIdx.x;
    //   int kernel_base = blockIdx.x * (IC * K * K) + ic * (K * K);
    //   for (int kh = 0; kh < K; kh++)
    //     for (int kw = 0; kw < K; kw++) {
    //       val += input[input_base + kh * (W) + kw] *
    //              weight[kernel_base + kh * (K) + kw];
    //     }
    // }
    // output[taskIdx] = val;

    // int taskIdx = blockIdx.y * gridDim.x * blockDim.y * blockDim.x
    //               + blockIdx.x * blockDim.y * blockDim.x
    //               + threadIdx.y * blockDim.x
    //               + threadIdx.x;

    __shared__ double input_sh[32][32];
    __shared__ double weight_sh[5][5];

    int H_OUT = H - (K - 1);
    int W_OUT = W - (K - 1);

    double val = bias[blockIdx.x];

    for (int ic = 0; ic < IC; ic++) {
      // load weights to shared memory
      if(threadIdx.y < K && threadIdx.x < K) {
        weight_sh[threadIdx.y][threadIdx.x] = weight[
          blockIdx.x * IC * K * K + ic * K * K + threadIdx.y * K + threadIdx.x];
      }
      __syncthreads();

      // load input to shared memory
      input_sh[threadIdx.y][threadIdx.x] = input[blockIdx.y * (IC * H * W) + ic * (H * W) + threadIdx.y * W + threadIdx.x];
      __syncthreads();

      if((threadIdx.y < H_OUT) && (threadIdx.x < W_OUT)) {
        for (int kh = 0; kh < K; kh++) {
          for (int kw = 0; kw < K; kw++) {
            val += input_sh[threadIdx.y + kh][threadIdx.x + kw] *
                   weight_sh[kh][kw];
          }
        }
      }
      __syncthreads();
    }

    if((threadIdx.y < H_OUT) && (threadIdx.x < W_OUT)) {
      int outIdx = blockIdx.y * gridDim.x * H_OUT * W_OUT
                    + blockIdx.x * H_OUT * W_OUT
                    + threadIdx.y * W_OUT
                    + threadIdx.x;
      output[outIdx] = val;
    }

}

__global__
void cuda_im2col(double* input, double* output_col, int H, int W, int K) {
  // blockIdx.y : mini-batch (b)
  // blockIdx.x : input Channel (ic)
  // threadIdx.y : Height (h), blockDim.y = H_OUT
  // threadIdx.x : Width (w), blockDim.x = W_OUT
  int b = blockIdx.y;
  int ic = blockIdx.x;
  int h = threadIdx.y;
  int w = threadIdx.x;
  int IC = gridDim.x;
  int H_OUT = blockDim.y;
  int W_OUT = blockDim.x;
  int C_base = ic * K * K;
  for(int p=0; p<K; p++) {
    for(int q=0; q<K; q++) {
      int y = C_base + p * K + q;
      int x = h * W_OUT + w;
      output_col[b * (IC * K * K) * (H_OUT * W_OUT) + y * (H_OUT * W_OUT) + x] =
        input[b * IC * H * W + ic * H * W + (h + p) * W + (w + q)];
    }
  }
}

__global__
void cuda_mm(double* input, double* output, double* weight, double* bias, int IC, int K) {
  // blockIdx.y : mini-batch (b)
  // blockIdx.x : output Channel (oc)
  // threadIdx.y : Height (h), blockDim.y = H_OUT
  // threadIdx.x : Width (w), blockDim.x = W_OUT
  int b = blockIdx.y;
  int oc = blockIdx.x;
  int h = threadIdx.y;
  int w = threadIdx.x;
  int OC = gridDim.x;
  int H_OUT = blockDim.y;
  int W_OUT = blockDim.x;
  // int K2 = K * K;
  int ICKK = IC * K * K;

  double val = bias[oc];
  // for(int ic = 0; ic < IC; ic++) {
  //   for(int k = 0; k < K2; k++) {
  //     val += weight[b * IC * OC * K2 + ic * OC * K2 + oc * K2 + k]
  //           * input[b * IC * K2 * H_OUT * W_OUT + ic * K2 * H_OUT * W_OUT + k * H_OUT * W_OUT + h * W_OUT + w];
  //   }
  // }
  for(int i = 0; i < ICKK; i++) {
    val += weight[oc * ICKK + i] * input[b * ICKK * (H_OUT * W_OUT) + i * (H_OUT * W_OUT) + h * W_OUT + w];
  }
  output[b * OC * H_OUT * W_OUT + oc * H_OUT * W_OUT + h * W_OUT + w] = val;
}

__global__
void cuda_relu(double* feature_map) {
  // for (int i = 0; i < size; i++) feature_map[i] = std::max(feature_map[i], 0.0);
  // blockIdx.x : [batch, channel]
  // threadIdx.x : [H, W]
  int taskIdx = blockIdx.x * blockDim.x + threadIdx.x;
  // if (feature_map[taskIdx] < 0.0)
  //   feature_map[taskIdx] = 0.0;
  feature_map[taskIdx] = fmax(feature_map[taskIdx], 0.0);
}

__global__
void cuda_pool(double* input, double* output) {
  // // Initilaize variable
  // int scale = 2;
  // int H_OUT = H / scale;
  // int W_OUT = W / scale;
  // // Max Pooling
  // for (int b = 0; b < B; b++)
  //   for (int c = 0; c < C; c++)
  //     for (int h = 0; h < H; h += 2)
  //       for (int w = 0; w < W; w += 2) {
  //         // Init values
  //         int input_base = b * (C * H * W) + c * (H * W) + h * (W) + w;
  //         int max_sh = 0;
  //         int max_sw = 0;
  //         double max_val = std::numeric_limits<double>::lowest();
  //         // Find maximum
  //         for (int sh = 0; sh < scale; sh++)
  //           for (int sw = 0; sw < scale; sw++) {
  //             double val = input[input_base + sh * (W) + sw];
  //             if (val - max_val > std::numeric_limits<double>::epsilon()) {
  //               max_val = val;
  //               max_sh = sh;
  //               max_sw = sw;
  //             }
  //           }
  //         // Set output with max value
  //         int output_index = b * (C * H_OUT * W_OUT) + c * (H_OUT * W_OUT) +
  //                            (h / 2) * W_OUT + (w / 2);
  //         output[output_index] = max_val;
  //       }
  // blockIdx.y : BATCH
  // blockIdx.x : Channel
  // threadIdx.y : output h
  // threadIdx.x : output w
  int BLKSIZE = blockDim.y * blockDim.x;
  // int taskIdx = blockIdx.y * gridDim.x * blockDim.y * blockDim.x
  //               + blockIdx.x * blockDim.y * blockDim.x
  //               + threadIdx.y * blockDim.x
  //               + threadIdx.x;
  // int taskIdx = blockIdx.y * gridDim.x * BLKSIZE
  //               + blockIdx.x * BLKSIZE
  //               + threadIdx.y * blockDim.x
  //               + threadIdx.x;
  int BLKSIZE2 = 4 * BLKSIZE;
  // int input_base = blockIdx.y * gridDim.x * (2*blockDim.y) * (2*blockDim.x)
  //               + blockIdx.x * (2*blockDim.y) * (2*blockDim.x)
  //               + (2*threadIdx.y) * (2*blockDim.x)
  //               + (2*threadIdx.x);
  int input_base = blockIdx.y * gridDim.x * BLKSIZE2
                + blockIdx.x * BLKSIZE2
                + (2*threadIdx.y) * (2*blockDim.x)
                + (2*threadIdx.x);
  double max_val = 0.0;
  for (int sh = 0; sh < 2; sh++)
    for (int sw = 0; sw < 2; sw++) {
      double val = input[input_base + sh * (2*blockDim.x) + sw];
      // if(val > max_val) {
      //   max_val = val;
      // }
      max_val = fmax(max_val, val);
    }
  // output[taskIdx] = max_val;
  output[blockIdx.y * gridDim.x * BLKSIZE + blockIdx.x * BLKSIZE
                + threadIdx.y * blockDim.x + threadIdx.x] = max_val;
}

__global__
void cuda_fc(double* input, double* output, double* weight, double* bias,
                    int IC) {
  // // Fully Connected
  // for (int b = 0; b < B; b++)
  //   for (int oc = 0; oc < OC; oc++) {
  //     output[b * OC + oc] = bias[oc];
  //     for (int ic = 0; ic < IC; ic++)
  //       output[b * OC + oc] += weight[oc * IC + ic] * input[b * IC + ic];
  //   }

  // blockIdx.x : BATCH
  // threadIdx.x : out_channel
  int taskIdx = blockIdx.x * blockDim.x + threadIdx.x;
  double val = bias[threadIdx.x];
  for(int ic=0; ic<IC; ic++) {
    val += weight[threadIdx.x * IC + ic] * input[blockIdx.x * IC + ic];
  }
  output[taskIdx] = val;
}

void LeNet5_cuda::predict(int batch) {
  // uint8_t* image;
  // image = new uint8_t[batch * IMG_SIZE];
  // size_t image_size = batch * input_size * input_size * input_channel;
  // hipMemcpy(image, d_image, image_size * sizeof(uint8_t),
  //            hipMemcpyDeviceToHost);
  /************************* Normalize **********************/
  dim3 DimGrid(input_channel * batch);
  dim3 DimBlock(input_size * input_size);
  normalize<<<DimGrid, DimBlock>>>(d_image, d_input);
  hipDeviceSynchronize();

  // hipMemcpy(input, d_input,
  //            batch * input_size * input_size * input_channel * sizeof(double),
  //            hipMemcpyDeviceToHost);

  // *********** Conv1 ***************//

  // DimGrid.y = batch; DimGrid.x = conv1_out_channel;
  // DimBlock.y = 28; //input_size - (conv1_kernel_size - 1);
  // DimBlock.x = 28; //input_size - (conv1_kernel_size - 1);
  // cuda_conv<<<DimGrid, DimBlock>>>(d_input, d_C1_feature_map, d_conv1_weight, d_conv1_bias, input_size,
  //     input_size, conv1_in_channel, conv1_kernel_size);
  // hipDeviceSynchronize();

  DimGrid.y = batch; DimGrid.x = conv1_out_channel;
  DimBlock.y = 28;
  DimBlock.x = 28;
  cuda_conv1<<<DimGrid, DimBlock>>>(d_input, d_C1_feature_map, d_conv1_weight,
      d_conv1_bias);
  hipDeviceSynchronize();

  // DimGrid.y = batch; DimGrid.x = conv1_out_channel;
  // DimBlock.y = 28;
  // DimBlock.x = 28;
  // cuda_conv1_sh<<<DimGrid, DimBlock>>>(d_input, d_C1_feature_map, d_conv1_weight, d_conv1_bias, batch, input_size,
  //     input_size, conv1_in_channel, conv1_out_channel, conv1_kernel_size);
  // hipDeviceSynchronize();

  // DimGrid.y = batch; DimGrid.x = conv1_in_channel;
  // DimBlock.y = input_size - (conv1_kernel_size - 1);
  // DimBlock.x = input_size - (conv1_kernel_size - 1);
  // cuda_im2col<<<DimGrid, DimBlock>>>(d_input, d_input_col, input_size, input_size, conv1_kernel_size);
  // hipDeviceSynchronize();
  // DimGrid.y = batch; DimGrid.x = conv1_out_channel;
  // DimBlock.y = input_size - (conv1_kernel_size - 1);
  // DimBlock.x = input_size - (conv1_kernel_size - 1);
  // cuda_mm<<<DimGrid, DimBlock>>>(d_input_col, d_C1_feature_map, d_conv1_weight,
  //    d_conv1_bias, conv1_in_channel, conv1_kernel_size);
  // hipDeviceSynchronize();

  // hipMemcpy(C1_feature_map, d_C1_feature_map,
  //            batch * conv1_out_channel * C1_size * C1_size * sizeof(double),
  //            hipMemcpyDeviceToHost);

  /**************** relu *****************/

  DimGrid.y = 1; DimGrid.x = batch * C1_channel;
  DimBlock.y = 1; DimBlock.x = C1_size * C1_size;
  cuda_relu<<<DimGrid, DimBlock>>>(d_C1_feature_map);
  hipDeviceSynchronize();

  // hipMemcpy(C1_feature_map, d_C1_feature_map,
  //            batch * conv1_out_channel * C1_size * C1_size * sizeof(double),
  //            hipMemcpyDeviceToHost);

/******************** Pool1 **********************/
  // MaxPool2d
  DimGrid.y = batch; DimGrid.x = C1_channel;
  DimBlock.y = C1_size / 2; DimBlock.x = C1_size / 2;
  cuda_pool<<<DimGrid, DimBlock>>>(d_C1_feature_map, d_S2_feature_map);
  hipDeviceSynchronize();

  // hipMemcpy(S2_feature_map, d_S2_feature_map,
  //            batch * C1_channel * (C1_size / 2) * (C1_size / 2) * sizeof(double),
  //            hipMemcpyDeviceToHost);

  /************************ Conv2 *****************************/

  // DimGrid.y = batch; DimGrid.x = conv2_out_channel;
  // DimBlock.y = 10; //S2_size - (conv2_kernel_size - 1);
  // DimBlock.x = 10; //S2_size - (conv2_kernel_size - 1);
  // cuda_conv<<<DimGrid, DimBlock>>>(d_S2_feature_map, d_C3_feature_map,
  //     d_conv2_weight, d_conv2_bias, S2_size,
  //     S2_size, conv2_in_channel, conv2_kernel_size);
  // hipDeviceSynchronize();

  DimGrid.y = batch; DimGrid.x = conv2_out_channel;
  DimBlock.y = 10; //S2_size - (conv2_kernel_size - 1);
  DimBlock.x = 10; //S2_size - (conv2_kernel_size - 1);
  cuda_conv2<<<DimGrid, DimBlock>>>(d_S2_feature_map, d_C3_feature_map,
      d_conv2_weight, d_conv2_bias);
  hipDeviceSynchronize();

  /****************************relu****************************/

  DimGrid.y = 1; DimGrid.x = batch * C3_channel;
  DimBlock.y = 1; DimBlock.x = C3_size * C3_size;
  cuda_relu<<<DimGrid, DimBlock>>>(d_C3_feature_map);
  hipDeviceSynchronize();

/*************************** Pool2 ****************************/
  // MaxPool2d
  DimGrid.y = batch; DimGrid.x = C3_channel;
  DimBlock.y = C3_size / 2; DimBlock.x = C3_size / 2;
  cuda_pool<<<DimGrid, DimBlock>>>(d_C3_feature_map, d_S4_feature_map);
  hipDeviceSynchronize();

  // hipMemcpy(S4_feature_map, d_S4_feature_map,
  //            batch * C3_channel * (C3_size / 2) * (C3_size / 2) * sizeof(double),
  //            hipMemcpyDeviceToHost);

/*************************** fc1 ***************************/
  // Linear
  DimGrid.y = 1; DimGrid.x = batch;
  DimBlock.y = 1; DimBlock.x = fc1_out_channel;
  cuda_fc<<<DimGrid, DimBlock>>>(d_S4_feature_map, d_C5_layer,
    d_fc1_weight, d_fc1_bias, fc1_in_channel);
  hipDeviceSynchronize();

  // hipMemcpy(C5_layer, d_C5_layer,
  //            batch * fc1_out_channel * sizeof(double),
  //            hipMemcpyDeviceToHost);

/*************************** relu ***********************/
  DimGrid.y = 1; DimGrid.x = batch;
  DimBlock.y = 1; DimBlock.x = C5_size;
  cuda_relu<<<DimGrid, DimBlock>>>(d_C5_layer);
  hipDeviceSynchronize();

/************************* fc2 ************************/
  // Linear
  DimGrid.y = 1; DimGrid.x = batch;
  DimBlock.y = 1; DimBlock.x = fc2_out_channel;
  cuda_fc<<<DimGrid, DimBlock>>>(d_C5_layer, d_F6_layer,
    d_fc2_weight, d_fc2_bias, fc2_in_channel);
  hipDeviceSynchronize();

/*************************** relu ***********************/
  DimGrid.y = 1; DimGrid.x = batch;
  DimBlock.y = 1; DimBlock.x = F6_size;
  cuda_relu<<<DimGrid, DimBlock>>>(d_F6_layer);
  hipDeviceSynchronize();

/****************************** fc3 *************************/
  // Linear
  DimGrid.y = 1; DimGrid.x = batch;
  DimBlock.y = 1; DimBlock.x = fc3_out_channel;
  cuda_fc<<<DimGrid, DimBlock>>>(d_F6_layer, d_output,
    d_fc3_weight, d_fc3_bias, fc3_in_channel);
  hipDeviceSynchronize();
  // hipMemcpy(d_output, output, sizeof(double) * output_size * batch,
  //            hipMemcpyHostToDevice);

    // TODO: Implement conv1
    // TODO: Implement relu
    // TODO: Implement pool1
    // TODO: Implement conv2
    // TODO: Implement relu
    // TODO: Implement pool2
    // TODO: Implement fc1
    // TODO: Implement relu
    // TODO: Implement fc2
    // TODO: Implement relu
    // TODO: Implement fc3

    /* NOTE: unless you want to make a major change to this class structure,
    *  you need to write your output to the device memory d_output
    *  so that classify() can handle the rest.
    */
}

void LeNet5_cuda::prepare_device_memory(uint8_t* image) {
  // Alloc Model Parameters
  hipMalloc((void**)&d_conv1_weight,
             sizeof(double) * conv1_in_channel * conv1_out_channel *
                 conv1_kernel_size * conv1_kernel_size);
  hipMalloc((void**)&d_conv1_bias, sizeof(double) * conv1_out_channel);
  hipMalloc((void**)&d_conv2_weight,
             sizeof(double) * conv2_in_channel * conv2_out_channel *
                 conv2_kernel_size * conv2_kernel_size);
  hipMalloc((void**)&d_conv2_bias, sizeof(double) * conv2_out_channel);
  hipMalloc((void**)&d_fc1_weight,
             sizeof(double) * fc1_in_channel * fc1_out_channel);
  hipMalloc((void**)&d_fc1_bias, sizeof(double) * fc1_out_channel);
  hipMalloc((void**)&d_fc2_weight,
             sizeof(double) * fc2_in_channel * fc2_out_channel);
  hipMalloc((void**)&d_fc2_bias, sizeof(double) * fc2_out_channel);
  hipMalloc((void**)&d_fc3_weight,
             sizeof(double) * fc3_in_channel * fc3_out_channel);
  hipMalloc((void**)&d_fc3_bias, sizeof(double) * fc3_out_channel);

  // Alloc Activations
  hipMalloc((void**)&d_image,
             sizeof(uint8_t) * batch * input_size * input_size * input_channel);
  hipMalloc((void**)&d_input,
             sizeof(double) * batch * input_channel * input_size * input_size);
  hipMalloc((void**)&d_C1_feature_map,
             sizeof(double) * batch * C1_channel * C1_size * C1_size);
  hipMalloc((void**)&d_S2_feature_map,
             sizeof(double) * batch * S2_channel * S2_size * S2_size);
  hipMalloc((void**)&d_C3_feature_map,
             sizeof(double) * batch * C3_channel * C3_size * C3_size);
  hipMalloc((void**)&d_S4_feature_map,
             sizeof(double) * batch * S4_channel * S4_size * S4_size);
  hipMalloc((void**)&d_C5_layer, sizeof(double) * batch * C5_size);
  hipMalloc((void**)&d_F6_layer, sizeof(double) * batch * F6_size);
  hipMalloc((void**)&d_output, sizeof(double) * batch * output_size);

  // hipMalloc((void**)&d_input_col,
  //            sizeof(double) * batch * input_channel * conv1_kernel_size * conv1_kernel_size * C1_size * C1_size);

  // Copy Parameters
  hipMemcpy(d_conv1_weight, conv1_weight,
             sizeof(double) * conv1_in_channel * conv1_out_channel *
                 conv1_kernel_size * conv1_kernel_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv1_bias, conv1_bias, sizeof(double) * conv1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv2_weight, conv2_weight,
             sizeof(double) * conv2_in_channel * conv2_out_channel *
                 conv2_kernel_size * conv2_kernel_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv2_bias, conv2_bias, sizeof(double) * conv2_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc1_weight, fc1_weight,
             sizeof(double) * fc1_in_channel * fc1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc1_bias, fc1_bias, sizeof(double) * fc1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc2_weight, fc2_weight,
             sizeof(double) * fc2_in_channel * fc2_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc2_bias, fc2_bias, sizeof(double) * fc2_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc3_weight, fc3_weight,
             sizeof(double) * fc3_in_channel * fc3_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc3_bias, fc3_bias, sizeof(double) * fc3_out_channel,
             hipMemcpyHostToDevice);
  // copy input image
  size_t image_size = batch * input_size * input_size * input_channel;
  hipMemcpy(d_image, image, image_size * sizeof(uint8_t),
             hipMemcpyHostToDevice);
}

void LeNet5_cuda::classify(int* predict, int batch) {
  // read logits back to cpu
  hipMemcpy(output, d_output, sizeof(double) * output_size * batch,
             hipMemcpyDeviceToHost);
  // Softmax
  softmax(output, predict, batch, output_size);
}

LeNet5_cuda::~LeNet5_cuda() {
  hipFree(d_conv1_weight);
  hipFree(d_conv2_weight);
  hipFree(d_conv1_bias);
  hipFree(d_conv2_bias);
  hipFree(d_fc1_weight);
  hipFree(d_fc2_weight);
  hipFree(d_fc3_weight);
  hipFree(d_fc1_bias);
  hipFree(d_fc2_bias);
  hipFree(d_fc3_bias);

  hipFree(d_image);
  hipFree(d_input);
  hipFree(d_C1_feature_map);
  hipFree(d_S2_feature_map);
  hipFree(d_C3_feature_map);
  hipFree(d_S4_feature_map);
  hipFree(d_C5_layer);
  hipFree(d_F6_layer);
  hipFree(d_output);
  hipFree(d_predict_cuda);

  // hipFree(d_input_col);
}
